#include "hip/hip_runtime.h"
#include "statarray.cuh"

#include "hip/hip_runtime.h"
#include ""

#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>

std::shared_ptr<polynomial> polynomial::operator+(const polynomial& other) const
{
	thrust::device_vector<float> d1 = this->coefficients;
	thrust::device_vector<float> d2 = other.coefficients;

	transform(d1.begin(), d1.end(), d2.begin(), d2.begin(), thrust::plus<float>());

	std::shared_ptr<polynomial> ret = std::make_shared<polynomial>(this->degree);

	thrust::copy(d2.begin(), d2.end(), ret->coefficients.begin());
	
	return ret;
}

std::shared_ptr<polynomial> polynomial::operator-(const polynomial& other) const
{
	thrust::device_vector<float> d1 = this->coefficients;
	thrust::device_vector<float> d2 = other.coefficients;

	transform(d1.begin(), d1.end(), d2.begin(), d2.begin(), thrust::minus<float>());

	std::shared_ptr<polynomial> ret = std::make_shared<polynomial>(this->degree);

	thrust::copy(d2.begin(), d2.end(), ret->coefficients.begin());

	return ret;
}
//
//std::shared_ptr<polynomial> polynomial::operator+=(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator-=(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator*(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator/(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::cut(int degree)
//{
//}


std::shared_ptr<statarray> statarray::operator[](const vector<int>& ids) const
{
	std::shared_ptr<statarray> ret;
	ret->reserve(ids.size());
	for (auto id : ids)
	{
		ret->push_back(this->at(id));
	}
	return ret;
}

std::shared_ptr<statarray> statarray::operator[](const statarray& predicate_vector) const
{
	std::shared_ptr<statarray> ret;
	ret->reserve(predicate_vector.size());
	auto i = this->begin();
	for(auto belongs : predicate_vector)
	{
		if(belongs)
		{
			ret->push_back(*i);
		}
		++i;
	}
	std::vector<float>(ret->begin(), ret->end()).swap(*ret);
	return ret;
}


std::shared_ptr<statarray> statarray::operator*(float f) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::multiplies<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator*(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::multiplies<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator+(float f) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::multiplies<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator+(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::plus<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator-(float f) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::minus<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator-(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::minus<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator/(float f) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::divides<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator/(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::divides<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

struct power_functor
{
	__host__ __device__
	float operator()(float a, float k) const
	{
		return powf(a, k);
	}
};

std::shared_ptr<statarray> statarray::operator^(float f) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), power_functor());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator^(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), power_functor());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator<(float f) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::less<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

//std::shared_ptr<polynomial> polynomial::operator+=(const polynomial& other)
//{
//
//}

std::shared_ptr<statarray> operator*(const float& f, const statarray& right)
{
	thrust::device_vector<float> d1 = right;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::multiplies<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(right.size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> operator+(const float& f, const statarray& right)
{
	thrust::device_vector<float> d1 = right;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::plus<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(right.size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}



std::shared_ptr<statarray> operator-(const float& f, const statarray& right)
{
	thrust::device_vector<float> d1 = right;
	thrust::constant_iterator<float> fiterator(f);

	transform(thrust::make_transform_iterator(d1.begin(), thrust::negate<float>()), 
		thrust::make_transform_iterator(d1.end(), thrust::negate<float>()),
		fiterator, 
		d1.begin(), 
		thrust::plus<float>()
	);

	std::shared_ptr<statarray> ret;
	ret->reserve(right.size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

struct mirror_div_func
{
	__host__ __device__
	float operator()(float a, float b) const
	{
		return b / a;
	}
};

std::shared_ptr<statarray> operator/(const float& f, const statarray& right)
{
	thrust::device_vector<float> d1 = right;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), mirror_div_func());

	std::shared_ptr<statarray> ret;
	ret->reserve(right.size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

struct mirror_pow_func
{
	__host__ __device__
		float operator()(float a, float b) const
	{
		return powf(b, a);
	}
};

std::shared_ptr<statarray> operator^(const float& f, const statarray& right)
{
	thrust::device_vector<float> d1 = right;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), mirror_pow_func());

	std::shared_ptr<statarray> ret;
	ret->reserve(right.size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator-() const
{
	thrust::device_vector<float> d1 = *this;

	transform(d1.begin(), d1.end(), d1.begin(), thrust::negate<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

//std::shared_ptr<polynomial> polynomial::operator-=(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator*(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator/(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::cut(int degree)
//{
//}

std::shared_ptr<statarray> operator<(const float& f, const statarray& right)
{
	return right > f;
}

std::shared_ptr<statarray> operator>(const float& f, const statarray& right)
{
	return right < f;
}

std::shared_ptr<statarray> operator<=(const float& f, const statarray& right)
{
	return right >= f;
}

std::shared_ptr<statarray> operator>=(const float& f, const statarray& right)
{
	return right <= f;
}

std::shared_ptr<statarray> operator==(const float& f, const statarray& right)
{
	return right == f;
}

std::shared_ptr<statarray> statarray::operator<(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::less<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator<=(float f) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::less_equal<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator<=(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::less_equal<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator>=(float f) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::greater_equal<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator>=(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::greater_equal<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator>(float f) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::greater<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator>(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::greater<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator==(float f) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::equal_to<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator==(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::equal_to<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator*=(float f)
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::multiplies<float>());

	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator*=(const statarray& other)
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::multiplies<float>());

	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator+=(float f)
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::plus<float>());

	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator+=(const statarray& other)
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::plus<float>());

	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator-=(float f)
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::minus<float>());

	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator-=(const statarray& other)
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::minus<float>());

	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator/=(float f)
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), thrust::divides<float>());

	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator/=(const statarray& other)
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::divides<float>());

	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator^=(float f)
{
	thrust::device_vector<float> d1 = *this;
	thrust::constant_iterator<float> fiterator(f);

	transform(d1.begin(), d1.end(), fiterator, d1.begin(), power_functor());

	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator^=(const statarray& other)
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), power_functor());

	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator|(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::logical_or<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator&(const statarray& other) const
{
	thrust::device_vector<float> d1 = *this;
	thrust::device_vector<float> d2 = other;

	transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::logical_and<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator!() const
{
	thrust::device_vector<float> d1 = *this;

	transform(d1.begin(), d1.end(), d1.begin(), thrust::logical_not<float>());

	std::shared_ptr<statarray> ret;
	ret->reserve(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::sample(int n) const
{
}

std::shared_ptr<statarray> statarray::sorted(sortorder order) const
{
}

std::shared_ptr<std::vector<int>> statarray::order(sortorder order) const
{
}

float statarray::sum() const
{
}

float statarray::mean() const
{
}

float statarray::mode() const
{
}

float statarray::iqm() const
{
}

float statarray::min() const
{
}

float statarray::max() const
{
}

float statarray::median() const
{
}

float statarray::lquart() const
{
}

float statarray::uquart() const
{
}

float statarray::quantile(float q) const
{
}

float statarray::iqr() const
{
}

float statarray::stdev() const
{
}

float statarray::variance() const
{
}

float statarray::skewness() const
{
}

float statarray::kurtosis() const
{
}

float statarray::covariance(const statarray& other) const
{
}

float statarray::correlation(const statarray& other) const
{
}

float statarray::harmonic_mean() const
{
}

float statarray::geometric_mean(int k) const
{
}

float statarray::generalized_mean(int k) const
{
}

float statarray::winsorized_mean(float fraction) const
{
}

float statarray::truncated_mean(float fraction) const
{
}

float statarray::weighted_arithmetic_mean(const vector<float>& weights) const
{
}

bool statarray::shapiro_wilk_test() const
{
}

bool statarray::t_test() const
{
}

std::shared_ptr<statarray> statarray::standardized() const
{
}

std::shared_ptr<std::vector<int>> statarray::histogram(int nbins) const
{
}

std::shared_ptr<polynomial> statarray::least_squares(const statarray& other) const
{
}

void statarray::to_csv(std::string filename) const
{
}

void statarray::from_csv(std::string filename) const
{
}

float correlation(const statarray& v1, const statarray& v2)
{
}

std::shared_ptr<polynomial> least_squares(const statarray& v1, const statarray& v2)
{
}

std::shared_ptr<std::vector<bool>> or(std::vector<bool>& v1, std::vector<bool>& v2)
{
}

std::shared_ptr<std::vector<bool>> and(std::vector<bool>& v1, std::vector<bool>& v2)
{
}

std::shared_ptr<std::vector<bool>> not(std::vector<bool>& v1, std::vector<bool>& v2)
{
}
