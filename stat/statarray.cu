#include "hip/hip_runtime.h"
#include "statarray.cuh"



std::shared_ptr<polynomial> polynomial::operator+(const polynomial& other) const
{
	thrust::device_vector<float> d1 = this->coefficients;
	thrust::device_vector<float> d2 = other.coefficients;

	transform(d1.begin(), d1.end(), d2.begin(), d2.begin(), thrust::plus<float>());

	std::shared_ptr<polynomial> ret = std::make_shared<polynomial>(this->degree);

	thrust::copy(d2.begin(), d2.end(), ret->coefficients.begin());
	
	return ret;
}

std::shared_ptr<polynomial> polynomial::operator-(const polynomial& other) const
{
	thrust::device_vector<float> d1 = this->coefficients;
	thrust::device_vector<float> d2 = other.coefficients;

	transform(d1.begin(), d1.end(), d2.begin(), d2.begin(), thrust::minus<float>());

	std::shared_ptr<polynomial> ret = std::make_shared<polynomial>(this->degree);

	thrust::copy(d2.begin(), d2.end(), ret->coefficients.begin());

	return ret;
}
//
//std::shared_ptr<polynomial> polynomial::operator+=(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator-=(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator*(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator/(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::cut(int degree)
//{
//}

struct rnorm_functor : thrust::unary_function<int, float>
{
	float mean, stdev;
	__host__ __device__
	rnorm_functor(float mean, float stdev) :mean(mean), stdev(stdev) {}

	__host__ __device__
	float operator()(const unsigned int n) const
	{
		thrust::default_random_engine rng;
		thrust::normal_distribution<float> dist(mean, stdev);
		rng.discard(n);

		return dist(rng);
	}
};

std::shared_ptr<statarray> statarray::rnorm(int n, float mean, float stdev)
{
	thrust::device_vector<float> dnumbers(n);
	thrust::counting_iterator<int> i(0);

	thrust::transform(i, i + n, dnumbers.begin(), rnorm_functor(mean, stdev));

	std::shared_ptr<statarray> ret = std::make_shared<statarray>();
	ret->resize(n);

	thrust::copy(dnumbers.begin(), dnumbers.end(), ret->begin());

	return ret;
}


struct runif_int_functor : thrust::unary_function<int, int>
{
	int a, b;
	__host__ __device__
		runif_int_functor(int a, int b) :a(a), b(b) {}

	__host__ __device__
		float operator()(const unsigned int n) const
	{
		thrust::default_random_engine rng;
		thrust::uniform_int_distribution<int> dist(a, b);
		rng.discard(n);

		return dist(rng);
	}
};

std::shared_ptr<statarray> statarray::randint(int n, int minvalue, int maxvalue)
{
	thrust::device_vector<float> dnumbers(n);
	thrust::counting_iterator<int> i(0);

	thrust::transform(i, i + n, dnumbers.begin(), runif_int_functor(minvalue, maxvalue));

	std::shared_ptr<statarray> ret = std::make_shared<statarray>();
	ret->resize(n);

	thrust::copy(dnumbers.begin(), dnumbers.end(), ret->begin());

	return ret;
}

struct runif_float_functor : thrust::unary_function<int, float>
{
	float a, b;
	__host__ __device__
		runif_float_functor(float a, float b) :a(a), b(b) {}

	__host__ __device__
		float operator()(const unsigned int n) const
	{
		thrust::default_random_engine rng;
		thrust::uniform_real_distribution<float> dist(a, b);
		rng.discard(n);

		return dist(rng);
	}
};

std::shared_ptr<statarray> statarray::randfloat(int n, float minvalue, float maxvalue)
{
	thrust::device_vector<float> dnumbers(n);
	thrust::counting_iterator<int> i(0);

	thrust::transform(i, i + n, dnumbers.begin(), runif_float_functor(minvalue, maxvalue));

	std::shared_ptr<statarray> ret = std::make_shared<statarray>();
	ret->resize(n);

	thrust::copy(dnumbers.begin(), dnumbers.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::range(int low_incl, int high_excl)
{
	std::shared_ptr<statarray> ret = std::make_shared<statarray>();
	ret->resize(high_excl - low_incl);
	thrust::device_vector<float> d(ret->size());
	thrust::sequence(d.begin(), d.end(), low_incl);
	thrust::copy(d.begin(), d.end(), ret->begin());
	return ret;
}


std::shared_ptr<statarray> statarray::rep(int nelements) const
{
	if(this->size() > nelements)
	{
		return std::make_shared<statarray>(this->begin(),this->begin()+(nelements-1));
	}
	else if(this->size() < nelements)
	{
		std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->begin(), this->end());
		ret->reserve(nelements);
		while(ret->size() < nelements)
		{
			ret->insert(ret->end(), ret->begin(), ret->end());
		}
		ret->resize(nelements);
		ret->shrink_to_fit();
		return ret;
	}
	return std::make_shared<statarray>(this->begin(), this->end());
}

std::shared_ptr<statarray> statarray::operator[](const vector<int>& ids) const
{
	std::shared_ptr<statarray> ret = std::make_shared<statarray>("");
	ret->reserve(ids.size());
	for (auto id : ids)
	{
		ret->push_back(this->at(id));
	}
	return ret;
}

std::shared_ptr<statarray> statarray::operator[](const statarray& predicate_vector) const
{
	std::shared_ptr<statarray> ret = std::make_shared<statarray>("");
	ret->reserve(predicate_vector.size());
	auto i = this->begin();
	for(auto belongs : predicate_vector)
	{
		if(belongs)
		{
			ret->push_back(*i);
		}
		++i;
	}
	std::vector<float>(ret->begin(), ret->end()).swap(*ret);
	return ret;
}

std::shared_ptr<statarray> statarray::operator*(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::multiplies<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("*").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator+(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::plus<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("+").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator-(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::minus<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("-").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator/(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::divides<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("/").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

struct power_functor
{
	__host__ __device__
	float operator()(float a, float k) const
	{
		return powf(a, k);
	}
};


std::shared_ptr<statarray> statarray::operator^(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), power_functor());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("^").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}



//std::shared_ptr<polynomial> polynomial::operator+=(const polynomial& other)
//{
//
//}


struct mirror_div_func
{
	__host__ __device__
	float operator()(float a, float b) const
	{
		return b / a;
	}
};

struct mirror_pow_func
{
	__host__ __device__
		float operator()(float a, float b) const
	{
		return powf(b, a);
	}
};

std::shared_ptr<statarray> statarray::operator-() const
{
	thrust::device_vector<float> d1 = *this;

	thrust::transform(d1.begin(), d1.end(), d1.begin(), thrust::negate<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(std::string("-").append(this->get_name()));
	ret->resize(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

float statarray::product() const
{
	thrust::device_vector<float> d = *this;

	return thrust::reduce(d.begin(), d.end(), 1, thrust::multiplies<float>());
}

std::shared_ptr<statarray> statarray::trim(float left, float right)
{
	return this->trim(round(left*this->size()), round(right*this->size()));
}

std::shared_ptr<statarray> statarray::trimmed(float left, float right) const
{
	return this->trimmed(round(left*this->size()), round(right*this->size()));
}

std::shared_ptr<statarray> statarray::trim(int left, int right)
{
	std::shared_ptr<statarray> trimmed = this->trimmed(left, right);
	thrust::swap(*this, *trimmed);
	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::trimmed(int left, int right) const
{
	std::shared_ptr<statarray> sorted = this->sorted();
	if (right < 0) right = left;

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append(".trimmed"));
	ret->resize(this->size() - left - right);
	thrust::copy(sorted->begin() + left, sorted->end() - right, ret->begin());

	return ret;
}

//std::shared_ptr<polynomial> polynomial::operator-=(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator*(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator/(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::cut(int degree)
//{
//}


std::shared_ptr<statarray> statarray::operator<(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::less<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("<").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator<=(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::less_equal<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("<=").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator>=(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::greater_equal<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append(">=").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator>(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::greater<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append(">").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator==(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::equal_to<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("==").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator*=(const statarray& other)
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::multiplies<float>());

	this->resize(size);
	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator+=(const statarray& other)
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::plus<float>());

	this->resize(size);
	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator-=(const statarray& other)
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::minus<float>());

	this->resize(size);
	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator/=(const statarray& other)
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::divides<float>());

	this->resize(size);
	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator^=(const statarray& other)
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), power_functor());

	this->resize(size);
	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator|(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::logical_or<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("|").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator&(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::logical_and<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("&").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator!() const
{
	thrust::device_vector<float> d1 = *this;

	thrust::transform(d1.begin(), d1.end(), d1.begin(), thrust::logical_not<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(std::string("1").append(this->get_name()));
	ret->resize(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::sample(int n) const
{
	auto temp = this->randint(n, 0, this->size() - 1);
	return (*this)[std::vector<int>(temp->begin(), temp->end())];
}

std::shared_ptr<statarray> statarray::sort(sortorder order)
{
	thrust::swap(*this, *this->sorted(order));
	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::sorted(sortorder order) const
{
	thrust::device_vector<float> d(this->size());
	thrust::copy(this->begin(), this->end(), d.begin());
	if(order == DESC)
	{
		thrust::sort(d.begin(), d.end(), thrust::greater<float>());
	}
	else
	{
		thrust::sort(d.begin(), d.end());
	}
	
	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append(".sorted"));
	ret->resize(this->size());

	thrust::copy(d.begin(), d.end(), ret->begin());

	return ret;
}

//std::shared_ptr<std::vector<int>> statarray::order(sortorder order) const
//{
//}

float statarray::sum() const
{
	thrust::device_vector<float> d(this->begin(), this->end());

	return thrust::reduce(d.begin(), d.end(), 0, thrust::plus<float>());
}

float statarray::mean() const
{
	return this->sum() / this->size();
}

float statarray::mode() const
{
	//TODO: implement
	return 0;
}

float statarray::iqm() const
{
	return this->trimmed(0.25F)->mean();
	
}

float statarray::min() const
{
	thrust::device_vector<float> d(this->begin(), this->end());

	return *thrust::min_element(d.begin(), d.end());
}

float statarray::max() const
{
	thrust::device_vector<float> d(this->begin(), this->end());

	return *thrust::max_element(d.begin(), d.end());
}

float statarray::median() const
{
	return this->quantile(0.5);
}

float statarray::lquart() const
{
	return this->quantile(0.25);
}

float statarray::uquart() const
{
	return this->quantile(0.75);
}

float statarray::quantile(float q) const
{
	q = std::min<float>(1, std::max<float>(0, q));
	std::shared_ptr<statarray> sorted = this->sorted();
	int before = floor(q * this->size());
	int after = floor((1 - q) * this->size());

	return ((*sorted)[before] + (*sorted)[this->size() - after - 1]) * 0.5;
}

float statarray::iqr() const
{
	return this->uquart() - this->lquart();
}

float statarray::expected_value() const
{
	return this->mean();
}

float statarray::stdev() const
{
	return sqrt(this->variance());
}

float statarray::variance() const
{
//	return (*this ^ 2)->expected_value() - pow(this->expected_value(), 2);
	return (*(*this - this->mean()) ^ 2)->sum() / (this->size() - 1);
}

float statarray::skewness() const
{
	return 3 * (this->mean() - this->median()) / this->stdev();
}

float statarray::kurtosis() const
{
	return (*(*this - this->expected_value()) ^ 4)->expected_value();
}

float statarray::covariance(const statarray& other) const
{
	return (*(*this - this->mean()) * *(other - other.mean()))->sum() / (this->size() - 1);
}

float statarray::correlation(const statarray& other) const
{
	return this->covariance(other) / (this->stdev() * other.stdev());
}

float statarray::harmonic_mean() const
{
	return float(this->size()) / ((*this) ^ (-1))->sum();
}

float statarray::geometric_mean() const
{
	return powf(this->product(), 1.0 / this->size());
}

float statarray::generalized_mean(float p) const
{
	return powf((*this ^ p)->sum() / this->size(), 1 / p);
}

float statarray::winsorized_mean(int nleft, int nright) const
{
	if (nright < 0) nright = nleft;
	float l = this->at(nleft);
	float r = this->at(this->size() - nright - 1);

	thrust::constant_iterator<float> il(l);
	thrust::constant_iterator<float> ir(r);

	statarray temp(this->begin(), this->end());

	thrust::copy_n(il, nleft, temp.begin());
	thrust::copy_n(ir, nright, temp.end() - nright);

	return temp.mean();
}

float statarray::winsorized_mean(float fraction_left, float fraction_right) const
{
	int nleft = floor(fraction_left * this->size());
	int nright = floor(fraction_right * this->size());

	return this->winsorized_mean(nleft, nright);
}

float statarray::truncated_mean(float fraction_left, float fraction_right) const
{
	return this->trimmed(fraction_left, fraction_right)->mean();
}

float statarray::truncated_mean(int nleft, int nright) const
{
	return this->trimmed(nleft, nright)->mean();
}

float statarray::weighted_arithmetic_mean(const vector<float>& weights) const
{
	thrust::device_vector<float> x = *this;
	thrust::device_vector<float> w = weights;

	thrust::transform(x.begin(), x.end(), w.begin(), x.begin(), thrust::multiplies<float>());
	return thrust::reduce(x.begin(), x.end(), 0, thrust::plus<float>()) / this->size();
}

bool statarray::shapiro_wilk_test() const
{
	//TODO: implement
	return 0;
}

bool statarray::t_test() const
{
	//TODO: implement
	return 0;
}

std::shared_ptr<statarray> statarray::standardized() const
{
	return *(*this - this->mean()) / this->stdev();
}

std::shared_ptr<statarray> statarray::winsorized(int nleft, int nright) const
{
	if (nright < 0) nright = nleft;
	float l = this->at(nleft);
	float r = this->at(this->size() - nright - 1);

	thrust::constant_iterator<float> il(l);
	thrust::constant_iterator<float> ir(r);

	statarray temp(this->begin(), this->end());

	thrust::copy_n(il, nleft, temp.begin());
	thrust::copy_n(ir, nright, temp.end() - nright);

	return std::make_shared<statarray>(temp);
}

std::shared_ptr<statarray> statarray::winsorized(float fraction_left, float fraction_right) const
{
	int nleft = floor(fraction_left * this->size());
	int nright = floor(fraction_right * this->size());

	return this->winsorized(nleft, nright);
}


struct histogram_functor : thrust::unary_function<float, float>
{
	float binspan;
	__host__ __device__
	histogram_functor(float binspan) :binspan(binspan) {}

	__host__ __device__
	float operator()(float val) 
	{
		return floor(val / binspan);
	}
};

std::shared_ptr<statarray> statarray::histogram(int nbins) const
{
	auto sorted = this->sorted();
	thrust::device_vector<int> dbins(nbins);
	thrust::device_vector<int> dbincounts(nbins);
	thrust::device_vector<float> dsorted(sorted->begin(), sorted->end());

	thrust::transform(dsorted.begin(), dsorted.end(), dsorted.begin(), histogram_functor((this->max() - this->min()) / this->size()));
	thrust::reduce_by_key(dsorted.begin(), dsorted.end(), thrust::constant_iterator<int>(1), dbins.begin(), dbincounts.begin(), thrust::equal_to<int>());

	auto ret = std::make_shared<statarray>();
	ret->set_name(this->get_name().append(".histogram"));
	ret->resize(nbins);
	thrust::copy(dbincounts.begin(), dbincounts.end(), ret->begin());
	return ret;
}

std::shared_ptr<polynomial> statarray::least_squares(const statarray& other) const
{
	auto size = std::max(this->size(), other.size());
	auto x = this->rep(size);
	auto y = other.rep(size);
	auto s = size;
	auto sx = x->sum();
	auto sy = y->sum();
	auto sxy = (*x * *y)->sum();
	auto sxx = (*x * *x)->sum();
	auto syy = (*y * *y)->sum();
	auto delta = s*sxx - sx*sx;

	float a = (s*sxy - sx*sy) / delta;
	float b = (sxx*sy - sx*sxy) / delta;

	std::vector<float> coeff(2);
	coeff[0] = b;
	coeff[1] = a;

	return std::make_shared<polynomial>(coeff);
}

void statarray::to_csv(std::string filename) const
{
	std::ofstream outcsv;
	outcsv.open(filename, std::ios::out | std::ios::trunc);
	outcsv << this->get_name() << "\n";
	for (auto value : *this)
	{
		outcsv << value << "\n";
	}
	outcsv.close();
}

std::shared_ptr<statarray> statarray::from_csv(std::string filename)
{
	std::shared_ptr<statarray> ret = std::make_shared<statarray>();
	std::ifstream incsv;
	std::string line;
	incsv.open(filename, std::ios::in);
	incsv >> ret->name;
	
	float temp;
	while(!incsv.eof())
	{
		incsv >> temp;
		ret->push_back(temp);
	}
	ret->pop_back();

	ret->shrink_to_fit();

	return ret;
}

float correlation(const statarray& v1, const statarray& v2)
{
	return v1.correlation(v2);
}

float covariance(const statarray& v1, const statarray& v2)
{
	return v1.covariance(v2);
}

std::shared_ptr<polynomial> least_squares(const statarray& v1, const statarray& v2)
{
	return v1.least_squares(v2);
}

//std::shared_ptr<statarray> statarray::transform(const statarray& v, const thrust::binary_function<float, float, float>& binary_functor) const
//{
//	thrust::device_vector<float> d1(this->begin(), this->end());
//	thrust::device_vector<float> d2(v.begin(), v.end());
//
//	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), binary_functor);
//
//	std::shared_ptr<statarray> ret = std::make_shared<statarray>();
//	ret->resize(this->size());
//
//	thrust::copy(d1.begin(), d1.end(), ret->begin());
//
//	return ret;
//}
//
//std::shared_ptr<statarray> statarray::transform(const thrust::unary_function<float, float>& unary_functor) const
//{
//	thrust::device_vector<float> d(this->begin(), this->end());
//
//	thrust::transform(d.begin(), d.end(), d.begin(),unary_functor);
//
//	std::shared_ptr<statarray> ret = std::make_shared<statarray>();
//	ret->resize(this->size());
//
//	thrust::copy(d.begin(), d.end(), ret->begin());
//
//	return ret;
//}
//
//std::shared_ptr<statarray> statarray::transform_modify(const statarray& v, const thrust::binary_function<float, float, float>& binary_functor)
//{
//	thrust::device_vector<float> d1(this->begin(), this->end());
//	thrust::device_vector<float> d2(v.begin(), v.end());
//
//	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), binary_functor);
//
//	thrust::copy(d1.begin(), d1.end(), this->begin());
//
//	return std::make_shared<statarray>(*this);
//}
//
//std::shared_ptr<statarray> statarray::transform_modify(const thrust::unary_function<float,float>& unary_functor)
//{
//	thrust::device_vector<float> d(this->begin(), this->end());
//
//	thrust::transform(d.begin(), d.end(), d.begin(), unary_functor);
//
//	thrust::copy(d.begin(), d.end(), this->begin());
//
//	return std::make_shared<statarray>(*this);
//}

std::shared_ptr<std::vector<bool>> or(std::vector<bool>& v1, std::vector<bool>& v2)
{
	thrust::device_vector<bool> d1 = v1;
	thrust::device_vector<bool> d2 = v2;

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::logical_or<bool>());

	std::shared_ptr<std::vector<bool>> ret;
	ret->resize(v1.size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<std::vector<bool>> and(std::vector<bool>& v1, std::vector<bool>& v2)
{
	thrust::device_vector<bool> d1 = v1;
	thrust::device_vector<bool> d2 = v2;

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::logical_and<bool>());

	std::shared_ptr<std::vector<bool>> ret;
	ret->resize(v1.size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<std::vector<bool>> not(std::vector<bool>& v)
{
	thrust::device_vector<bool> d = v;

	thrust::transform(d.begin(), d.end(), d.begin(), thrust::logical_not<bool>());

	std::shared_ptr<std::vector<bool>> ret;
	ret->resize(v.size());

	thrust::copy(d.begin(), d.end(), ret->begin());

	return ret;
}

void print(const statarray& v)
{
	std::cout << "Statarray " << v.get_name() << "\n";
	std::cout << "Size: " << v.size() << " elements\n";
	int count = 0;
	for(int i = 0; i < v.size(); i++)
	{
		std::cout << i << "\t" << *(v.begin()+i) << std::endl;
		if (++count > 20)
			break;
	}
	if (count > 20)
		printf("...\n");
	printf("\n");
}


