#include "hip/hip_runtime.h"
#include "statarray.cuh"

#include "hip/hip_runtime.h"
#include ""

#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <algorithm>

std::shared_ptr<polynomial> polynomial::operator+(const polynomial& other) const
{
	thrust::device_vector<float> d1 = this->coefficients;
	thrust::device_vector<float> d2 = other.coefficients;

	transform(d1.begin(), d1.end(), d2.begin(), d2.begin(), thrust::plus<float>());

	std::shared_ptr<polynomial> ret = std::make_shared<polynomial>(this->degree);

	thrust::copy(d2.begin(), d2.end(), ret->coefficients.begin());
	
	return ret;
}

std::shared_ptr<polynomial> polynomial::operator-(const polynomial& other) const
{
	thrust::device_vector<float> d1 = this->coefficients;
	thrust::device_vector<float> d2 = other.coefficients;

	transform(d1.begin(), d1.end(), d2.begin(), d2.begin(), thrust::minus<float>());

	std::shared_ptr<polynomial> ret = std::make_shared<polynomial>(this->degree);

	thrust::copy(d2.begin(), d2.end(), ret->coefficients.begin());

	return ret;
}
//
//std::shared_ptr<polynomial> polynomial::operator+=(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator-=(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator*(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator/(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::cut(int degree)
//{
//}


//std::shared_ptr<statarray> statarray::rnorm(int n)
//{
//}
//
//std::shared_ptr<statarray> statarray::randint(int n, int minvalue, int maxvalue)
//{
//}
//
//std::shared_ptr<statarray> statarray::randfloat(int n, float minvalue, float maxvalue)
//{
//}


std::shared_ptr<statarray> statarray::rep(int nelements) const
{
	if(this->size() > nelements)
	{
		return std::make_shared<statarray>(this->begin(),this->begin()+(nelements-1));
	}
	else if(this->size() < nelements)
	{
		std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->begin(), this->end());
		ret->reserve(nelements);
		while(ret->size() < nelements)
		{
			ret->insert(ret->end(), ret->begin(), ret->end());
		}
		ret->resize(nelements);
		ret->shrink_to_fit();
		return ret;
	}
	return std::make_shared<statarray>(this->begin(), this->end());
}

std::shared_ptr<statarray> statarray::operator[](const vector<int>& ids) const
{
	std::shared_ptr<statarray> ret = std::make_shared<statarray>("");
	ret->reserve(ids.size());
	for (auto id : ids)
	{
		ret->push_back(this->at(id));
	}
	return ret;
}

std::shared_ptr<statarray> statarray::operator[](const statarray& predicate_vector) const
{
	std::shared_ptr<statarray> ret = std::make_shared<statarray>("");
	ret->reserve(predicate_vector.size());
	auto i = this->begin();
	for(auto belongs : predicate_vector)
	{
		if(belongs)
		{
			ret->push_back(*i);
		}
		++i;
	}
	std::vector<float>(ret->begin(), ret->end()).swap(*ret);
	return ret;
}

std::shared_ptr<statarray> statarray::operator*(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::multiplies<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("*").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator+(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::plus<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("+").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator-(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::minus<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("-").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator/(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::divides<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("/").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

struct power_functor
{
	__host__ __device__
	float operator()(float a, float k) const
	{
		return powf(a, k);
	}
};


std::shared_ptr<statarray> statarray::operator^(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), power_functor());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("^").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}



//std::shared_ptr<polynomial> polynomial::operator+=(const polynomial& other)
//{
//
//}


struct mirror_div_func
{
	__host__ __device__
	float operator()(float a, float b) const
	{
		return b / a;
	}
};

struct mirror_pow_func
{
	__host__ __device__
		float operator()(float a, float b) const
	{
		return powf(b, a);
	}
};

std::shared_ptr<statarray> statarray::operator-() const
{
	thrust::device_vector<float> d1 = *this;

	thrust::transform(d1.begin(), d1.end(), d1.begin(), thrust::negate<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(std::string("-").append(this->get_name()));
	ret->resize(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

float statarray::product() const
{
	thrust::device_vector<float> d = *this;

	return thrust::reduce(d.begin(), d.end(), 1, thrust::multiplies<float>());
}

//std::shared_ptr<polynomial> polynomial::operator-=(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator*(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::operator/(const polynomial& other)
//{
//}
//
//std::shared_ptr<polynomial> polynomial::cut(int degree)
//{
//}


std::shared_ptr<statarray> statarray::operator<(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::less<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("<").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator<=(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::less_equal<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("<=").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator>=(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::greater_equal<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append(">=").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator>(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::greater<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append(">").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator==(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::equal_to<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("==").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator*=(const statarray& other)
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::multiplies<float>());

	this->resize(size);
	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator+=(const statarray& other)
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::plus<float>());

	this->resize(size);
	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator-=(const statarray& other)
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::minus<float>());

	this->resize(size);
	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator/=(const statarray& other)
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::divides<float>());

	this->resize(size);
	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator^=(const statarray& other)
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), power_functor());

	this->resize(size);
	thrust::copy(d1.begin(), d1.end(), this->begin());

	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::operator|(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::logical_or<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("|").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator&(const statarray& other) const
{
	int size = std::max(this->size(), other.size());
	thrust::device_vector<float> d1 = *this->rep(size);
	thrust::device_vector<float> d2 = *other.rep(size);

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::logical_and<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(this->get_name().append("&").append(other.get_name()));
	ret->resize(size);

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<statarray> statarray::operator!() const
{
	thrust::device_vector<float> d1 = *this;

	thrust::transform(d1.begin(), d1.end(), d1.begin(), thrust::logical_not<float>());

	std::shared_ptr<statarray> ret = std::make_shared<statarray>(std::string("1").append(this->get_name()));
	ret->resize(this->size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

//std::shared_ptr<statarray> statarray::sample(int n) const
//{
//}

std::shared_ptr<statarray> statarray::sort(sortorder order)
{
	thrust::swap(*this, *this->sorted(order));
	return std::make_shared<statarray>(*this);
}

std::shared_ptr<statarray> statarray::sorted(sortorder order) const
{
	//TODO: implement
	return nullptr;
}

//std::shared_ptr<std::vector<int>> statarray::order(sortorder order) const
//{
//}
//
float statarray::sum() const
{
	thrust::device_vector<float> d(this->begin(), this->end());

	return thrust::reduce(d.begin(), d.end(), 0, thrust::plus<float>());
}

float statarray::mean() const
{
	return this->sum() / this->size();
}
//
//float statarray::mode() const
//{
//}
//
//float statarray::iqm() const
//{
//}

float statarray::min() const
{
	thrust::device_vector<float> d(this->begin(), this->end());

	return *thrust::min_element(d.begin(), d.end());
}

float statarray::max() const
{
	thrust::device_vector<float> d(this->begin(), this->end());

	return *thrust::max_element(d.begin(), d.end());
}

float statarray::median() const
{
	return this->quantile(0.5);
}

float statarray::lquart() const
{
	return this->quantile(0.25);
}

float statarray::uquart() const
{
	return this->quantile(0.75);
}

float statarray::quantile(float q) const
{
	q = std::min<float>(1, std::max<float>(0, q));
	std::shared_ptr<statarray> sorted = this->sorted();
	int before = floor(q * this->size());
	int after = floor((1 - q) * this->size());

	return ((*sorted)[before] + (*sorted)[this->size() - after - 1]) * 0.5;
}

float statarray::iqr() const
{
	return this->uquart() - this->lquart();
}

float statarray::expected_value() const
{
	return this->mean();
}

float statarray::stdev() const
{
	return sqrt(this->variance());
}

float statarray::variance() const
{
	return (*this ^ 2)->expected_value() - pow(this->expected_value(), 2);
}

float statarray::skewness() const
{
	return 3 * (this->mean() - this->median()) / this->stdev();
}

float statarray::kurtosis() const
{
	return (*(*this - this->expected_value()) ^ 4)->expected_value();
}

//float statarray::covariance(const statarray& other) const
//{
//}
//
//float statarray::correlation(const statarray& other) const
//{
//}

float statarray::harmonic_mean() const
{
	return float(this->size()) / ((*this) ^ (-1))->sum();
}

float statarray::geometric_mean() const
{
	return powf(this->product(), 1.0 / this->size());
}

//float statarray::generalized_mean(int k) const
//{
//}
//
//float statarray::winsorized_mean(float fraction) const
//{
//}
//
//float statarray::truncated_mean(float fraction) const
//{
//}
//
//float statarray::weighted_arithmetic_mean(const vector<float>& weights) const
//{
//}
//
//bool statarray::shapiro_wilk_test() const
//{
//}
//
//bool statarray::t_test() const
//{
//}
//
//std::shared_ptr<statarray> statarray::standardized() const
//{
//}
//
//std::shared_ptr<std::vector<int>> statarray::histogram(int nbins) const
//{
//}
//
//std::shared_ptr<polynomial> statarray::least_squares(const statarray& other) const
//{
//}
//
//void statarray::to_csv(std::string filename) const
//{
//}
//
//void statarray::from_csv(std::string filename) const
//{
//}
//
//float correlation(const statarray& v1, const statarray& v2)
//{
//}
//
//std::shared_ptr<polynomial> least_squares(const statarray& v1, const statarray& v2)
//{
//}

//std::shared_ptr<statarray> statarray::transform(const statarray& v, const thrust::binary_function<float, float, float>& binary_functor) const
//{
//	thrust::device_vector<float> d1(this->begin(), this->end());
//	thrust::device_vector<float> d2(v.begin(), v.end());
//
//	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), binary_functor);
//
//	std::shared_ptr<statarray> ret = std::make_shared<statarray>();
//	ret->resize(this->size());
//
//	thrust::copy(d1.begin(), d1.end(), ret->begin());
//
//	return ret;
//}
//
//std::shared_ptr<statarray> statarray::transform(const thrust::unary_function<float, float>& unary_functor) const
//{
//	thrust::device_vector<float> d(this->begin(), this->end());
//
//	thrust::transform(d.begin(), d.end(), d.begin(),unary_functor);
//
//	std::shared_ptr<statarray> ret = std::make_shared<statarray>();
//	ret->resize(this->size());
//
//	thrust::copy(d.begin(), d.end(), ret->begin());
//
//	return ret;
//}
//
//std::shared_ptr<statarray> statarray::transform_modify(const statarray& v, const thrust::binary_function<float, float, float>& binary_functor)
//{
//	thrust::device_vector<float> d1(this->begin(), this->end());
//	thrust::device_vector<float> d2(v.begin(), v.end());
//
//	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), binary_functor);
//
//	thrust::copy(d1.begin(), d1.end(), this->begin());
//
//	return std::make_shared<statarray>(*this);
//}
//
//std::shared_ptr<statarray> statarray::transform_modify(const thrust::unary_function<float,float>& unary_functor)
//{
//	thrust::device_vector<float> d(this->begin(), this->end());
//
//	thrust::transform(d.begin(), d.end(), d.begin(), unary_functor);
//
//	thrust::copy(d.begin(), d.end(), this->begin());
//
//	return std::make_shared<statarray>(*this);
//}

std::shared_ptr<std::vector<bool>> or(std::vector<bool>& v1, std::vector<bool>& v2)
{
	thrust::device_vector<bool> d1 = v1;
	thrust::device_vector<bool> d2 = v2;

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::logical_or<bool>());

	std::shared_ptr<std::vector<bool>> ret;
	ret->resize(v1.size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<std::vector<bool>> and(std::vector<bool>& v1, std::vector<bool>& v2)
{
	thrust::device_vector<bool> d1 = v1;
	thrust::device_vector<bool> d2 = v2;

	thrust::transform(d1.begin(), d1.end(), d2.begin(), d1.begin(), thrust::logical_and<bool>());

	std::shared_ptr<std::vector<bool>> ret;
	ret->resize(v1.size());

	thrust::copy(d1.begin(), d1.end(), ret->begin());

	return ret;
}

std::shared_ptr<std::vector<bool>> not(std::vector<bool>& v)
{
	thrust::device_vector<bool> d = v;

	thrust::transform(d.begin(), d.end(), d.begin(), thrust::logical_not<bool>());

	std::shared_ptr<std::vector<bool>> ret;
	ret->resize(v.size());

	thrust::copy(d.begin(), d.end(), ret->begin());

	return ret;
}

void print(const statarray& v)
{
	std::cout << "Statarray " << v.get_name() << "\n";
	std::cout << "Size: " << v.size() << "\n";
	for (auto value : v)
	{
		std::cout << value << std::endl;
	}
	printf("\n");
}
